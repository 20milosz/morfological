#include "hip/hip_runtime.h"
#include "morfological.h"

#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime_api.h"//-I$(NVCUDASAMPLES_ROOT)/common/inc
#include "hip/hip_runtime.h"
#include ""

void copy(Matrix structuringElement)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(structuringElements), structuringElement.elements, strucElDim*strucElDim * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
}

void createStructuringElement(Matrix structuringElement)
{
	int i;
	for (int column = 0; column < strucElDim; column++)
	{
		for (int row = 0; row < strucElDim; row++)
		{
			i = column + strucElDim*row;
			if (row == strucElDim / 2 || column == strucElDim / 2)
			{
				structuringElement.elements[i] = (uint8_t)1;
			}
			else
			{
				structuringElement.elements[i] = (uint8_t)0;
			}
		}
	}
}
/*
__global__ void dilatation_cuda3(Matrix A, Matrix B, Matrix structuringElement)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (column == 0 && row == 0)printf("sf%d\n", structuringElement.elements[1]);

	if (column < A.numColumns && row < A.numRows)
	{

		int subMatrix[strucElDim*strucElDim];
		for (int i = 0; i < strucElDim*strucElDim; i++)
		{
			subMatrix[i] = 0;
		}
		int index = row * A.numColumns + column;
		int CValue = 0;
		int left = (column > 0 ? -strucElDim / 2 : 0);
		int right = (column < A.numColumns - 1 ? strucElDim / 2 : 0);
		int up = (row > 0 ? -strucElDim / 2 : 0);
		int down = (row < A.numRows - 1 ? strucElDim / 2 : 0);
		for (int i = up; i <= down; i++)
		{
			for (int j = left; j <= right; j++)
			{

				subMatrix[j + strucElDim / 2 + strucElDim * (i + strucElDim / 2)] = A.elements[index + j + i*A.numColumns];
			}
		}

		for (int i = 0; i < strucElDim*strucElDim; i++)
		{
			if (structuringElement.elements[i] * subMatrix[i] == 1)
				CValue = 1;
		}
		B.elements[index] = CValue;

	}
}

__global__ void dilatation_cuda2(Matrix A, Matrix result, Matrix structuringElement)
{

	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (column < A.numColumns && row < A.numRows)
	{
		uint8_t subMatrix[strucElDim*strucElDim];
		for (int i = 0; i < strucElDim*strucElDim; i++)
		{
			subMatrix[i] = 0;
		}
		int index;
		uint8_t CValue;

		index = row * A.numColumns + column;
		CValue = 0;

		for (int i = 0; i < strucElDim; i++)
		{
			for (int j = 0; j < strucElDim; j++)
			{
				if ((column - j >= strucElDim / 2) && (row - i >= strucElDim / 2) && (column + j <= A.numColumns - strucElDim / 2) && (row + i <= A.numRows - strucElDim / 2))
				{
					subMatrix[j + strucElDim * i] = A.elements[index + j - strucElDim / 2 + (i - strucElDim / 2)*A.numColumns];
				}
				else
				{
					subMatrix[j + strucElDim * i] = 0;
				}
			}
		}

		for (int i = 0; i < strucElDim*strucElDim; i++)
		{
			if (structuringElement.elements[i] * subMatrix[i] == 1)
				CValue = 1;
		}
	}
}


*/
__global__ void dilatation_cuda(Matrix A, Matrix result)
{
	int column = threadIdx.x+strucElDim/2;
	int row = threadIdx.y+strucElDim/2;

	printf("sdf\n");

	__shared__ uint8_t dilTile[(blockD+strucElDim-1)*(blockD+strucElDim-1)];
	
	dilTile[threadIdx.x + blockDim.x*threadIdx.y] = A.elements[threadIdx.x + A.numColumns*threadIdx.y + blockIdx.x*blockD + A.numColumns*blockD*blockIdx.y];
	__syncthreads();

	if (column < blockDim.x-strucElDim/2 && row < blockDim.y-strucElDim/2)
	{
		uint8_t subMatrix[strucElDim*strucElDim];
		for (int i = 0; i < strucElDim*strucElDim; i++)
		{
			subMatrix[i] = 0;
		}
		int index;
		uint8_t CValue;

		index = row * blockDim.x + column;
		CValue = 0;

		for (int i = -(strucElDim/2); i < strucElDim/2; i++)
		{
			for (int j = -(strucElDim/2); j < strucElDim/2; j++)
			{			
				subMatrix[j+strucElDim/2 + strucElDim * (i+strucElDim/2)] = dilTile[index + j + i*blockDim.x];
			}
		}

		for (int i = 0; i < strucElDim*strucElDim; i++)
		{
			if (structuringElements[i] * subMatrix[i] == 1)
				CValue = 1;
		}


		result.elements[threadIdx.x + strucElDim/2 + A.numColumns*(threadIdx.y+strucElDim/2) + blockIdx.x*blockD + A.numColumns*blockD*blockIdx.y] = CValue;
		

	}
	__syncthreads();

}


Matrix* dilatation(Matrix A, Matrix structuringElement)
{
		
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrix(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	verifyHostAllocation(*result);
	Matrix d_A;
	Matrix d_structuringElement;
	Matrix d_result;
	createDeviceMatrix(&d_A, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_structuringElement, strucElDim, strucElDim, strucElDim*strucElDim* sizeof(uint8_t));
	createDeviceMatrix(&d_result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	checkCudaErrors(hipMemcpy(d_A.elements, A.elements, A.numColumns*A.numRows * sizeof(uint8_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_structuringElement.elements, structuringElement.elements, strucElDim*strucElDim * sizeof(uint8_t), hipMemcpyHostToDevice));

	dim3 threads1(blockD + strucElDim - 1, blockD + strucElDim - 1);
	//dim3 threads(blockD + strucElDim + 100, blockD + strucElDim + 100);
	dim3 grid1(A.numColumns / blockD, A.numRows / blockD);
	printf("sfsdf\n");
	dilatation_cuda <<<grid1, threads1 >>> (d_A, d_result);
	checkCudaErrors(hipMemcpy(result->elements, d_result.elements, A.numColumns*A.numRows * sizeof(uint8_t), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_A.elements));
	checkCudaErrors(hipFree(d_structuringElement.elements));
	checkCudaErrors(hipFree(d_result.elements));

	return result;
}

Matrix* erosion(Matrix A, Matrix structuringElement)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrix(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	verifyHostAllocation(*result);
	uint8_t subMatrix[strucElDim*strucElDim];
	for (int i = 0; i < strucElDim*strucElDim; i++)
	{
		subMatrix[i] = 1;
	}
	int index;
	uint8_t CValue;
	for (int row = 0; row < A.numRows; row++)
	{
		for (int column = 0; column < A.numColumns; column++)
		{
			index = row * A.numColumns + column;
			CValue = 1;
			for (int i = 0; i < strucElDim; i++)
			{
				for (int j = 0; j < strucElDim; j++)
				{
					if ((column - j >= strucElDim / 2) && (row - i >= strucElDim / 2) && (column + j <= A.numColumns - strucElDim / 2) && (row + i <= A.numRows - strucElDim / 2))
					{
						subMatrix[j + strucElDim * i] = A.elements[index + j - strucElDim / 2 + (i - strucElDim / 2)*A.numColumns];
					}
					else
					{
						subMatrix[j + strucElDim * i] = 1;
					}
				}
			}

			for (int i = 0; i < strucElDim*strucElDim; i++)
			{
				if (structuringElement.elements[i] == 1 && subMatrix[i] == 0)
					CValue = 0;
			}
			result->elements[index] = CValue;
		}
	}
	return result;
}

__global__ void complement_cuda(Matrix A, Matrix B, Matrix result)
{
	int column = threadIdx.x + blockIdx.x*blockDim.x;
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	if (column < A.numColumns && row < A.numRows)
	{
		int index = row * A.numColumns + column;
		result.elements[index] = A.elements[index] * B.elements[index];

	}

}


Matrix* complement(Matrix A, Matrix B)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrix(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	verifyHostAllocation(*result);
	Matrix d_A;
	Matrix d_B;
	Matrix d_result;
	createDeviceMatrix(&d_A, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_B, B.numRows, B.numColumns, B.numColumns*B.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	checkCudaErrors(hipMemcpy(d_A.elements, A.elements, A.numColumns*A.numRows * sizeof(uint8_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_B.elements, B.elements, B.numColumns*B.numRows * sizeof(uint8_t), hipMemcpyHostToDevice));

	dim3 threads(blockD + strucElDim - 1, blockD + strucElDim - 1);
	dim3 grid((A.numColumns + threads.x-1) / threads.x, (A.numRows+threads.y-1) / threads.y);
	complement_cuda << <grid, threads >> > (d_A, d_B, d_result);
	checkCudaErrors(hipMemcpy(result->elements, d_result.elements, A.numColumns*A.numRows * sizeof(uint8_t), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_A.elements));
	checkCudaErrors(hipFree(d_B.elements));
	checkCudaErrors(hipFree(d_result.elements));

	return result;
}

Matrix* negation(Matrix A)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrix(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	verifyHostAllocation(*result);
	int index;
	for (int row = 0; row < A.numRows; row++)
	{
		for (int column = 0; column < A.numColumns; column++)
		{
			index = row * A.numColumns + column;
			result->elements[index] = (A.elements[index] ? 0 : 1);

		}
	}
	return result;
}



Matrix* opening(Matrix A, Matrix structuringElement)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	Matrix* resultErosion = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrixNoAllocation(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(resultErosion, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	resultErosion = erosion(A, structuringElement);
	result = dilatation(*resultErosion, structuringElement);
	free(resultErosion->elements);
	free(resultErosion);
	return result;
}

Matrix* closing(Matrix A, Matrix structuringElement)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	Matrix* resultDilatation = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrixNoAllocation(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(resultDilatation, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	resultDilatation = dilatation(A, structuringElement);
	result = erosion(*resultDilatation, structuringElement);
	free(resultDilatation->elements);
	free(resultDilatation);
	return result;
}
int checkIfEqual(Matrix A, Matrix B)
{
	int isEqual = 1;
	for (int i = 0; i < A.numRows*A.numColumns; i++)
	{
		if (A.elements[i] != B.elements[i])
		{
			isEqual = 0;
		}
	}
	return isEqual;
}

Matrix* reconstruction_cuda(Matrix mask, Matrix marker)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrix(result, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));

	Matrix d_mask;
	Matrix d_marker1;
	Matrix d_marker2;
	Matrix d_resultDil;
	createDeviceMatrix(&d_mask, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_marker1, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_marker2, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_resultDil, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	checkCudaErrors(hipMemcpy(d_mask.elements, mask.elements, mask.numColumns*mask.numRows * sizeof(uint8_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_marker1.elements, marker.elements, mask.numColumns*mask.numRows * sizeof(uint8_t), hipMemcpyHostToDevice));
	dim3 threadsDil(blockD + strucElDim - 1, blockD + strucElDim - 1);
	dim3 gridDil(mask.numColumns / blockD, mask.numRows / blockD);
	dim3 threadsComp(blockD + strucElDim - 1, blockD + strucElDim - 1);
	dim3 gridComp((mask.numColumns + threadsComp.x - 1) / threadsComp.x, (mask.numRows + threadsComp.y - 1) / threadsComp.y);
	
	
	for (int i = 0; i < 10; i++)
	{
		dilatation_cuda <<< gridDil, threadsDil >>> (d_marker1, d_resultDil);
		complement_cuda <<< gridComp, threadsComp >>> (d_resultDil, d_mask, d_marker2);
		dilatation_cuda <<< gridDil, threadsDil >>> (d_marker2, d_resultDil);
		complement_cuda <<< gridComp, threadsComp >>> (d_resultDil, d_mask, d_marker1);
	}
	
	checkCudaErrors(hipMemcpy(result->elements, d_marker1.elements, d_marker1.numColumns*d_marker1.numRows * sizeof(uint8_t), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_mask.elements));
	checkCudaErrors(hipFree(d_marker1.elements));
	checkCudaErrors(hipFree(d_marker2.elements));
	checkCudaErrors(hipFree(d_resultDil.elements));
	return result;
}

Matrix* reconstruction(Matrix mask, Matrix marker, Matrix structuringElement)
{

	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	Matrix* resultDil = (Matrix*)malloc(sizeof(Matrix));
	Matrix* marker1 = (Matrix*)malloc(sizeof(Matrix));
	Matrix* marker2 = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrixNoAllocation(result, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(resultDil, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(marker1, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(marker2, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	marker1 = &marker;
	resultDil = dilatation(*marker1, structuringElement);
	marker2 = complement(*resultDil, mask);


	marker1 = marker2;
	free(resultDil->elements);
	free(resultDil);
	resultDil = dilatation(*marker1, structuringElement);
	marker2 = complement(*resultDil, mask);


	while (!checkIfEqual(*marker1, *marker2))
	{
		free(marker1->elements);
		free(marker1);
		marker1 = marker2;
		free(resultDil->elements);
		free(resultDil);
		resultDil = dilatation(*marker1, structuringElement);
		marker2 = complement(*resultDil, mask);
	}
	free(marker1->elements);
	free(marker1);

	return marker2;
}


Matrix* openingByReconstruction(Matrix A, Matrix structuringElement)
{

	Matrix* resultEr = (Matrix*)malloc(sizeof(Matrix));
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrixNoAllocation(resultEr, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	resultEr = erosion(A, structuringElement);
	result = reconstruction_cuda(A, *resultEr);
	free(resultEr->elements);
	free(resultEr);
	return result;
}


