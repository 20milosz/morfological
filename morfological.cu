#include "hip/hip_runtime.h"
#include "morfological.h"

#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime_api.h"//-I$(NVCUDASAMPLES_ROOT)/common/inc
#include "hip/hip_runtime.h"
#include ""

void copy(Matrix structuringElement)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(structuringElements), structuringElement.elements, strucElDim*strucElDim * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
}

void createStructuringElement(Matrix structuringElement)
{
	int i;
	for (int column = 0; column < strucElDim; column++)
	{
		for (int row = 0; row < strucElDim; row++)
		{
			i = column + strucElDim*row;
			if (row == strucElDim / 2 || column == strucElDim / 2)
			{
				structuringElement.elements[i] = (uint8_t)1;
			}
			else
			{
				structuringElement.elements[i] = (uint8_t)0;
			}
		}
	}
}

__global__ void erosion_cuda(Matrix A, Matrix result)
{
	int column = threadIdx.x + strucElDim / 2;
	int row = threadIdx.y + strucElDim / 2;

	__shared__ uint8_t dilTile[(blockD + strucElDim - 1)*(blockD + strucElDim - 1)];

	dilTile[threadIdx.x + blockDim.x*threadIdx.y] = A.elements[threadIdx.x + A.numColumns*threadIdx.y + blockIdx.x*blockD + A.numColumns*blockD*blockIdx.y];
	__syncthreads();

	if (column < blockDim.x - strucElDim / 2 && row < blockDim.y - strucElDim / 2)
	{
		uint8_t subMatrix[strucElDim*strucElDim];
		int index;
		uint8_t CValue;

		index = row * blockDim.x + column;
		CValue = 1;
		
		for (int i = -(strucElDim / 2); i <= strucElDim / 2; i++)
		{
			for (int j = -(strucElDim / 2); j <= strucElDim / 2; j++)
			{
				subMatrix[j + strucElDim / 2 + strucElDim * (i + strucElDim / 2)] = dilTile[index + j + i*blockDim.x];
			}
		}
		for (int i = 0; i < strucElDim*strucElDim; i++)
		{
			if (structuringElements[i] == 1 && subMatrix[i] == 0)
				CValue = 0;
		}
		result.elements[threadIdx.x + strucElDim / 2 + A.numColumns*(threadIdx.y + strucElDim / 2) + blockIdx.x*blockD + A.numColumns*blockD*blockIdx.y] = CValue;
	}
	__syncthreads();
}

__global__ void dilatation_cuda(Matrix A, Matrix result)
{
	int column = threadIdx.x + strucElDim / 2;
	int row = threadIdx.y + strucElDim / 2;

	__shared__ uint8_t dilTile[(blockD + strucElDim - 1)*(blockD + strucElDim - 1)];

	dilTile[threadIdx.x + blockDim.x*threadIdx.y] = A.elements[threadIdx.x + A.numColumns*threadIdx.y + blockIdx.x*blockD + A.numColumns*blockD*blockIdx.y];
	__syncthreads();

	if (column < blockDim.x - strucElDim / 2 && row < blockDim.y - strucElDim / 2)
	{
		uint8_t subMatrix[strucElDim*strucElDim];
		int index;
		uint8_t CValue;

		index = row * blockDim.x + column;
		CValue = 0;

		for (int i = -(strucElDim / 2); i <= strucElDim / 2; i++)
		{
			for (int j = -(strucElDim / 2); j <= strucElDim / 2; j++)
			{
				subMatrix[j + strucElDim / 2 + strucElDim * (i + strucElDim / 2)] = dilTile[index + j + i*blockDim.x];
			}
		}
		for (int i = 0; i < strucElDim*strucElDim; i++)
		{
			if (structuringElements[i] * subMatrix[i] == 1)
				CValue = 1;
		}
		result.elements[threadIdx.x + strucElDim / 2 + A.numColumns*(threadIdx.y + strucElDim / 2) + blockIdx.x*blockD + A.numColumns*blockD*blockIdx.y] = CValue;
	}
	__syncthreads();
}


Matrix* dilatation(Matrix A)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrix(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	verifyHostAllocation(*result);
	Matrix d_A;
	Matrix d_result;
	createDeviceMatrix(&d_A, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	checkCudaErrors(hipMemcpy(d_A.elements, A.elements, A.numColumns*A.numRows * sizeof(uint8_t), hipMemcpyHostToDevice));
	dim3 threads1(blockD + strucElDim - 1, blockD + strucElDim - 1);
	dim3 grid1(A.numColumns / blockD, A.numRows / blockD);
	dilatation_cuda <<<grid1, threads1 >>> (d_A, d_result);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(result->elements, d_result.elements, A.numColumns*A.numRows * sizeof(uint8_t), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_A.elements));
	checkCudaErrors(hipFree(d_result.elements));
	return result;
}

Matrix* erosion(Matrix A)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrix(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	verifyHostAllocation(*result);
	Matrix d_A;
	Matrix d_result;
	createDeviceMatrix(&d_A, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	checkCudaErrors(hipMemcpy(d_A.elements, A.elements, A.numColumns*A.numRows * sizeof(uint8_t), hipMemcpyHostToDevice));
	dim3 threads1(blockD + strucElDim - 1, blockD + strucElDim - 1);
	dim3 grid1(A.numColumns / blockD, A.numRows / blockD);
	erosion_cuda << <grid1, threads1 >> > (d_A, d_result);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(result->elements, d_result.elements, A.numColumns*A.numRows * sizeof(uint8_t), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_A.elements));
	checkCudaErrors(hipFree(d_result.elements));
	return result;
}
__global__ void complement_cuda(Matrix A, Matrix B, Matrix result)
{
	int column = threadIdx.x + blockIdx.x*blockDim.x;
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	if (column < A.numColumns && row < A.numRows)
	{
		int index = row * A.numColumns + column;
		result.elements[index] = A.elements[index] * B.elements[index];

	}

}

Matrix* complement(Matrix A, Matrix B)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrix(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	verifyHostAllocation(*result);
	Matrix d_A;
	Matrix d_B;
	Matrix d_result;
	createDeviceMatrix(&d_A, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_B, B.numRows, B.numColumns, B.numColumns*B.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	checkCudaErrors(hipMemcpy(d_A.elements, A.elements, A.numColumns*A.numRows * sizeof(uint8_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_B.elements, B.elements, B.numColumns*B.numRows * sizeof(uint8_t), hipMemcpyHostToDevice));

	dim3 threads(blockD + strucElDim - 1, blockD + strucElDim - 1);
	dim3 grid((A.numColumns + threads.x-1) / threads.x, (A.numRows+threads.y-1) / threads.y);
	complement_cuda << <grid, threads >> > (d_A, d_B, d_result);
	checkCudaErrors(hipMemcpy(result->elements, d_result.elements, A.numColumns*A.numRows * sizeof(uint8_t), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_A.elements));
	checkCudaErrors(hipFree(d_B.elements));
	checkCudaErrors(hipFree(d_result.elements));

	return result;
}

Matrix* negation(Matrix A)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrix(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	verifyHostAllocation(*result);
	int index;
	for (int row = 0; row < A.numRows; row++)
	{
		for (int column = 0; column < A.numColumns; column++)
		{
			index = row * A.numColumns + column;
			result->elements[index] = (A.elements[index] ? 0 : 1);

		}
	}
	return result;
}

Matrix* opening(Matrix A)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	Matrix* resultErosion = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrixNoAllocation(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(resultErosion, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	resultErosion = erosion(A);
	result = dilatation(*resultErosion);
	free(resultErosion->elements);
	free(resultErosion);
	return result;
}

Matrix* closing(Matrix A)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	Matrix* resultDilatation = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrixNoAllocation(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(resultDilatation, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	resultDilatation = dilatation(A);
	result = erosion(*resultDilatation);
	free(resultDilatation->elements);
	free(resultDilatation);
	return result;
}
/*
__global__ int checkIfEqual_cuda(Matrix A, Matrix B)
{
	int column = threadIdx.x + strucElDim / 2;
	int row = threadIdx.y + strucElDim / 2;

	__shared__ uint8_t dilTile[(blockD + strucElDim - 1)*(blockD + strucElDim - 1)];
	if (A.elements[threadIdx.x + A.numColumns*threadIdx.y + blockIdx.x*blockD + A.numColumns*blockD*blockIdx.y] != B.elements[threadIdx.x + A.numColumns*threadIdx.y + blockIdx.x*blockD + A.numColumns*blockD*blockIdx.y])
		dilTile[threadIdx.x + blockDim.x*threadIdx.y] = 1;
	else
		dilTile[threadIdx.x + blockDim.x*threadIdx.y] = 0;
	__syncthreads();
	int tablica[1024];

	if (column < blockDim.x - strucElDim / 2 && row < blockDim.y - strucElDim / 2)
	{
		index = row * blockDim.x + column;
		dilTile[index]=
	}
	__syncthreads();
	return 1;
}
*/


int checkIfEqual(Matrix A, Matrix B)
{
	int isEqual = 1;
	for (int i = 0; i < A.numRows*A.numColumns; i++)
	{
		if (A.elements[i] != B.elements[i])
		{
			isEqual = 0;
		}
	}
	return isEqual;
}

Matrix* reconstruction_cuda(Matrix mask, Matrix marker)
{
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrix(result, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));

	Matrix d_mask;
	Matrix d_marker1;
	Matrix d_marker2;
	Matrix d_resultDil;
	createDeviceMatrix(&d_mask, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_marker1, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_marker2, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createDeviceMatrix(&d_resultDil, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	checkCudaErrors(hipMemcpy(d_mask.elements, mask.elements, mask.numColumns*mask.numRows * sizeof(uint8_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_marker1.elements, marker.elements, mask.numColumns*mask.numRows * sizeof(uint8_t), hipMemcpyHostToDevice));
	dim3 threadsDil(blockD + strucElDim - 1, blockD + strucElDim - 1);
	dim3 gridDil(mask.numColumns / blockD, mask.numRows / blockD);
	dim3 threadsComp(blockD + strucElDim - 1, blockD + strucElDim - 1);
	dim3 gridComp((mask.numColumns + threadsComp.x - 1) / threadsComp.x, (mask.numRows + threadsComp.y - 1) / threadsComp.y);
	
	int isEqual=1;
	for (int i = 0; i < 10; i++)
	//while(!checkIfEqual(d_marker1, d_marker2))
	{
		dilatation_cuda <<< gridDil, threadsDil >>> (d_marker1, d_resultDil);
		complement_cuda <<< gridComp, threadsComp >>> (d_resultDil, d_mask, d_marker2);
		dilatation_cuda <<< gridDil, threadsDil >>> (d_marker2, d_resultDil);
		complement_cuda <<< gridComp, threadsComp >>> (d_resultDil, d_mask, d_marker1);
	}
	
	checkCudaErrors(hipMemcpy(result->elements, d_marker1.elements, d_marker1.numColumns*d_marker1.numRows * sizeof(uint8_t), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_mask.elements));
	checkCudaErrors(hipFree(d_marker1.elements));
	checkCudaErrors(hipFree(d_marker2.elements));
	checkCudaErrors(hipFree(d_resultDil.elements));
	return result;
}

Matrix* reconstruction(Matrix mask, Matrix marker)
{

	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	Matrix* resultDil = (Matrix*)malloc(sizeof(Matrix));
	Matrix* marker1 = (Matrix*)malloc(sizeof(Matrix));
	Matrix* marker2 = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrixNoAllocation(result, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(resultDil, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(marker1, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(marker2, mask.numRows, mask.numColumns, mask.numColumns*mask.numRows * sizeof(uint8_t));
	marker1 = &marker;
	resultDil = dilatation(*marker1);
	marker2 = complement(*resultDil, mask);


	marker1 = marker2;
	free(resultDil->elements);
	free(resultDil);
	resultDil = dilatation(*marker1);
	marker2 = complement(*resultDil, mask);


	while (!checkIfEqual(*marker1, *marker2))
	{
		free(marker1->elements);
		free(marker1);
		marker1 = marker2;
		free(resultDil->elements);
		free(resultDil);
		resultDil = dilatation(*marker1);
		marker2 = complement(*resultDil, mask);
	}
	free(marker1->elements);
	free(marker1);

	return marker2;
}


Matrix* openingByReconstruction(Matrix A)
{

	Matrix* resultEr = (Matrix*)malloc(sizeof(Matrix));
	Matrix* result = (Matrix*)malloc(sizeof(Matrix));
	createHostMatrixNoAllocation(resultEr, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	createHostMatrixNoAllocation(result, A.numRows, A.numColumns, A.numColumns*A.numRows * sizeof(uint8_t));
	resultEr = erosion(A);
	result = reconstruction_cuda(A, *resultEr);
	free(resultEr->elements);
	free(resultEr);
	return result;
}